
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <algorithm>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

// Function to load vector from file
template <typename T>
void load_vector(std::vector<T>& vec, const std::string& filename)
{
    std::ifstream in_file(filename, std::ios::binary);
    if(!in_file)
    {
        std::cerr << "Cannot open the file: " << filename << std::endl;
        return;
    }

    size_t size;
    in_file.read(reinterpret_cast<char*>(&size), sizeof(size));
    vec.resize(size);
    in_file.read(reinterpret_cast<char*>(vec.data()), vec.size() * sizeof(T));
    in_file.close();
}

// Compute SpMV on CPU
std::vector<double> spmv_cpu(const std::vector<double>& values, 
                             const std::vector<int>& indices, 
                             const std::vector<int>& offsets, 
                             const std::vector<double>& x,
                             int n_constraints)
{
    // Initialize output vector with zeros
    std::vector<double> y(n_constraints, 0.0);
    
    // Ensure offsets size matches n_constraints + 1 for valid CSR representation
    if (offsets.size() != n_constraints + 1) {
        std::cerr << "Invalid CSR representation: offsets size does not match number of constraints.\n";
        return y;
    }

    auto start_time = std::chrono::high_resolution_clock::now();

    // Loop over each row
    for (int i = 0; i < n_constraints; ++i) {
        // Loop over non-zeros in the current row
        for (int j = offsets[i]; j < offsets[i + 1]; ++j) {
            // Perform multiplication and accumulate results
            y[i] += values[j] * x[indices[j]];
        }
    }

    return y;
}

std::vector<double> spmv_gpu(const double* d_values,
                             const int* d_indices,
                             const int* d_offsets,
                             const double* d_x,
                             hipsparseHandle_t& handle,
                             int n_variables,
                             int n_constraints)
{
    constexpr double alpha = 1.0;
    constexpr double beta = 0.0;

    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, n_constraints, n_variables, n_constraints,
                      (void*)d_offsets, (void*)d_indices, (void*)d_values,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    hipsparseDnVecDescr_t vecX;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n_variables, (void*)d_x, HIP_R_64F));

    // Vector used only for output as beta == 0
    double* d_y;
    CHECK_CUDA(hipMalloc(&d_y, n_constraints * sizeof(double)));
    hipsparseDnVecDescr_t vecY;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, n_constraints, (void*)d_y, HIP_R_64F));
    
    size_t bufferSize = 0;
    void* dBuffer;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_CSR_ALG2, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F, 
                 HIPSPARSE_SPMV_CSR_ALG2, dBuffer));

    std::vector<double> y(n_constraints);
    CHECK_CUDA(hipMemcpy(y.data(), d_y, n_constraints * sizeof(double), hipMemcpyDeviceToHost));
    
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    
    return y;
}

int main() {
    int major_version, minor_version;
    cusparseGetProperty(libraryPropertyType_t::MAJOR_VERSION, &major_version);
    cusparseGetProperty(libraryPropertyType_t::MINOR_VERSION, &minor_version);
    std::cout << "cuSparse version = " << major_version << "." << minor_version << std::endl;

    constexpr int n_variables = 17680;
    constexpr int n_constraints = 69608;

    // Load data
    std::vector<double> values, x;
    std::vector<int> indices, offsets;

    load_vector(values, "A_values");
    load_vector(x, "X");
    load_vector(offsets, "A_offsets");
    load_vector(indices, "A_indices");

    // Transfer data to device
    double* d_values;
    double* d_x;
    int* d_indices;
    int* d_offsets;
    CHECK_CUDA(hipMalloc(&d_values, values.size() * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_x, x.size() * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_indices, indices.size() * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_offsets, offsets.size() * sizeof(int)));

    CHECK_CUDA(hipMemcpy(d_values, values.data(), values.size() * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_indices, indices.data(), indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice));

    // Create cuSPARSE handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    
    // Compute SpMV on GPU
    std::vector<double> y_gpu = spmv_gpu(d_values, d_indices, d_offsets, d_x, handle, n_variables, n_constraints);
    std::vector<double> y_cpu = spmv_cpu(values, indices, offsets, x, n_constraints);
    
    // Compare the results
    double l2_diff = 0.0;
    int count_diff = 0;
    for (size_t i = 0; i < y_cpu.size(); ++i) {
        double diff = y_cpu[i] - y_gpu[i];
        l2_diff += diff * diff;  // add the square of the difference
        if (std::abs(diff) > 0.1) {
            ++count_diff;  // increment the counter
        }
    }
    l2_diff = std::sqrt(l2_diff);  // take the square root to complete the L2 norm computation
    std::cout << "L2 difference between CPU and custom GPU results: " << l2_diff << std::endl;
    std::cout << "Number of values with absolute difference > 0.1: " << count_diff << std::endl;

    // Clean up
    CHECK_CUDA(hipFree(d_values));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_indices));
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}